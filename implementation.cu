#include "hip/hip_runtime.h"
#include "implementation.h"

#include "stdio.h"
#define THEADS_PER_BLOCK 128


void printSubmissionInfo()
{
    // This will be published in the leaderboard on piazza
    // Please modify this field with something interesting
    char nick_name[] = "wobuxiangxie";

    // Please fill in your information (for marking purposes only)
    char student_first_name[] = "Ziyuan";
    char student_last_name[] = "Wang";
    char student_student_number[] = "1003968931";

    // Printing out team information
    printf("*******************************************************************************************************\n");
    printf("Submission Information:\n");
    printf("\tnick_name: %s\n", nick_name);
    printf("\tstudent_first_name: %s\n", student_first_name);
    printf("\tstudent_last_name: %s\n", student_last_name);
    printf("\tstudent_student_number: %s\n", student_student_number);
}


// add the parital sum back to the output for each element
__global__ void add_partial_sum(const int32_t* partial_sum, int32_t* output) {
    if (blockIdx.x >= 1) {
        unsigned global_index = threadIdx.x + blockIdx.x * blockDim.x;
        output[global_index] += partial_sum[blockIdx.x - 1];
    }
}

__global__ void scan(const int32_t* input, int32_t* output, int32_t* partial_sum, int size) {
    unsigned global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (global_index < size) {
        // copy to a shared memory and perform read/write on shared memory instead of global memory
        __shared__ int32_t per_block_result[THEADS_PER_BLOCK];
        per_block_result[threadIdx.x] = input[global_index];
        __syncthreads();

        for (int j = 1; j < THEADS_PER_BLOCK; j *= 2) {
            int32_t prev = 0;
            if (threadIdx.x >= j) {
                prev = per_block_result[threadIdx.x - j];
            }
            // there is a write after read, so need sychronization!!
            __syncthreads();
            if (threadIdx.x >= j) {
                per_block_result[threadIdx.x] += prev;
            }
            __syncthreads();
        }

        // record in partial sum if is last element in block
        if (threadIdx.x == THEADS_PER_BLOCK - 1) {
            partial_sum[blockIdx.x] = per_block_result[THEADS_PER_BLOCK - 1];
        }
        output[global_index] = per_block_result[threadIdx.x];
    }
}


/**
 * Implement your CUDA inclusive scan here. Feel free to add helper functions, kernels or allocate temporary memory.
 * However, you must not modify other files. CAUTION: make sure you synchronize your kernels properly and free all
 * allocated memory.
 *
 * @param d_input: input array on device
 * @param d_output: output array on device
 * @param size: number of elements in the input array
 */
void implementation(const int32_t *d_input, int32_t *d_output, size_t size) {
    unsigned num_blocks = ceil((float)size / THEADS_PER_BLOCK);

    int32_t* partial_sum;
    hipMalloc((void**) &partial_sum, num_blocks * sizeof(int32_t));
    hipDeviceSynchronize();
    
    // fill output with perblcok prefix sum and partial sum
    scan <<< num_blocks, THEADS_PER_BLOCK >>> (d_input, d_output, partial_sum, size);
    hipDeviceSynchronize();

    if (num_blocks > 1) {
        // recusive calling scan to calculate prefix sum of the partial sum, until all prefix sum fits in one block
        implementation(partial_sum, partial_sum, num_blocks);
        // add partial sum
        add_partial_sum <<< num_blocks, THEADS_PER_BLOCK >>> (partial_sum, d_output);
    }
    hipDeviceSynchronize();
}
